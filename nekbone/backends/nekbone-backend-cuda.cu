#include "hip/hip_runtime.h"
#include "nekbone-backend.h"

static uint initialized = 0;

#define check_error(FNAME, LINE, CALL, ERR_T, SUCCES, GET_ERR, OP)             \
  {                                                                            \
    ERR_T result_ = (CALL);                                                    \
    if (result_ != SUCCES) {                                                   \
      const char *msg = GET_ERR(result_);                                      \
      nekbone_error("%s:%d CUDA %s failure: %s.\n", FNAME, LINE, OP, msg);     \
    }                                                                          \
  }

#define check_driver(call)                                                     \
  check_error(__FILE__, __LINE__, call, hipError_t, hipSuccess,              \
              hipGetErrorName, "driver");

static scalar *d_r, *d_x, *d_z, *d_p, *d_w;
static scalar *d_wrk, *wrk;
static scalar *d_c, *d_g, *d_D;
static uint *d_gs_off, *d_gs_idx;
static const size_t local_size = 512;

static void cuda_mem_init(const struct nekbone_t *nekbone) {
  nekbone_debug(nekbone->verbose,
                "cuda_mem_init: copy problem data to device ... ");

  const uint n = nekbone_get_local_dofs(nekbone);

  // Allocate device buffers and copy problem data to device.
  check_driver(hipMalloc(&d_r, n * sizeof(scalar)));
  check_driver(hipMalloc(&d_x, n * sizeof(scalar)));
  check_driver(hipMalloc(&d_z, n * sizeof(scalar)));
  check_driver(hipMalloc(&d_p, n * sizeof(scalar)));
  check_driver(hipMalloc(&d_w, n * sizeof(scalar)));

  // Copy multiplicity array.
  check_driver(hipMalloc(&d_c, n * sizeof(scalar)));
  check_driver(
      hipMemcpy(d_c, nekbone->c, n * sizeof(scalar), hipMemcpyHostToDevice));

  // Copy geometric factors and derivative matrix.
  check_driver(hipMalloc(&d_g, 6 * n * sizeof(scalar)));
  check_driver(hipMemcpy(d_g, nekbone->g, 6 * n * sizeof(scalar),
                          hipMemcpyHostToDevice));

  check_driver(hipMalloc(&d_D, nekbone->nx1 * nekbone->nx1 * sizeof(scalar)));
  check_driver(hipMemcpy(d_D, nekbone->D,
                          nekbone->nx1 * nekbone->nx1 * sizeof(scalar),
                          hipMemcpyHostToDevice));

  // Copy gather-scatter offsets and indices.
  check_driver(hipMalloc(&d_gs_off, (nekbone->gs_n + 1) * sizeof(uint)));
  check_driver(hipMemcpy(d_gs_off, nekbone->gs_off,
                          (nekbone->gs_n + 1) * sizeof(uint),
                          hipMemcpyHostToDevice));

  check_driver(
      hipMalloc(&d_gs_idx, nekbone->gs_off[nekbone->gs_n] * sizeof(uint)));
  check_driver(hipMemcpy(d_gs_idx, nekbone->gs_idx,
                          nekbone->gs_off[nekbone->gs_n] * sizeof(uint),
                          hipMemcpyHostToDevice));

  // Work array.
  wrk = nekbone_calloc(scalar, n);
  check_driver(hipMalloc(&d_wrk, n * sizeof(scalar)));

  nekbone_debug(nekbone->verbose, "done.\n");
}

#define unifiedDeviceSynchronize hipDeviceSynchronize
#define unifiedMemcpy hipMemcpy
#define unifiedMemcpyDeviceToHost hipMemcpyDeviceToHost
#include "nekbone-backend-unified-cuda-hip.h"
#undef unifiedDeviceSynchronize
#undef unifiedMemcpy
#undef unifiedMemcpyDeviceToHost

static void cuda_init(const struct nekbone_t *nekbone) {
  if (initialized)
    return;
  nekbone_debug(nekbone->verbose, "cuda_init: initializing cuda backend ...\n");

  int num_devices = 0;
  check_driver(hipGetDeviceCount(&num_devices));
  if (nekbone->device >= (uint)num_devices) {
    nekbone_error("cuda_init: Invalid device id %d, only %d devices available.",
                  nekbone->device, num_devices);
  }

  check_driver(hipSetDeviceFlags(hipDeviceMapHost));
  check_driver(hipFree(0));

  cuda_mem_init(nekbone);

  initialized = 1;
  nekbone_debug(nekbone->verbose, "cuda_init: done.\n");
}

static scalar cuda_run(const struct nekbone_t *nekbone, const scalar *r) {
  if (!initialized)
    nekbone_error("cuda_run: cuda backend is not initialized.\n");

  const uint n = nekbone_get_local_dofs(nekbone);
  nekbone_debug(nekbone->verbose, "cuda_run: ... n=%u\n", n);

  clock_t t0 = clock();

  // Copy rhs to device buffer.
  check_driver(hipMemcpy(d_r, r, n * sizeof(scalar), hipMemcpyHostToDevice));

  scalar pap = 0;
  scalar rtz1 = 1, rtz2 = 0;

  // Zero out the solution.
  zero(d_x, n);

  // Apply Dirichlet BCs to RHS.
  mask(d_r, n);

  // Run CG on the device.
  scalar rnorm = sqrt(glsc3(d_r, d_c, d_r, n));
  scalar r0 = rnorm;
  for (uint i = 0; i < nekbone->max_iter; ++i) {
    // Preconditioner (which is just a copy for now).
    copy(d_z, d_r, n);

    rtz2 = rtz1;
    rtz1 = glsc3(d_r, d_c, d_z, n);

    scalar beta = rtz1 / rtz2;
    if (i == 0)
      beta = 0;
    add2s1(d_p, d_z, beta, n);

    ax(d_w, d_p, d_g, d_D, nekbone->nelt, nekbone->nx1);
    gs(d_w, d_gs_off, d_gs_idx, nekbone->gs_n);
    add2s2(d_w, d_p, 0.1, n);
    mask(d_w, n);

    pap = glsc3(d_w, d_c, d_p, n);

    scalar alpha = rtz1 / pap;
    add2s2(d_x, d_p, alpha, n);
    add2s2(d_r, d_w, -alpha, n);

    scalar rtr = glsc3(d_r, d_c, d_r, n);
    rnorm = sqrt(rtr);
    nekbone_debug(nekbone->verbose, "cuda_run: iteration %d, rnorm = %e\n", i,
                  rnorm);
  }

  check_driver(hipDeviceSynchronize());
  clock_t t1 = clock();

  nekbone_debug(nekbone->verbose, "cuda_run: done.\n");
  nekbone_debug(nekbone->verbose, "cuda_run: iterations = %d.\n",
                nekbone->max_iter);
  nekbone_debug(nekbone->verbose, "cuda_run: residual = %e %e.\n", r0, rnorm);

  return ((double)t1 - t0) / CLOCKS_PER_SEC;
}

static void cuda_finalize(void) {
  if (!initialized)
    return;

  check_driver(hipFree(d_r));
  check_driver(hipFree(d_x));
  check_driver(hipFree(d_z));
  check_driver(hipFree(d_p));
  check_driver(hipFree(d_w));
  check_driver(hipFree(d_c));
  check_driver(hipFree(d_g));
  check_driver(hipFree(d_D));
  check_driver(hipFree(d_gs_off));
  check_driver(hipFree(d_gs_idx));
  check_driver(hipFree(d_wrk));
  nekbone_free(&wrk);

  initialized = 0;
}

NEKBONE_INTERN void nekbone_cuda_init(void) {
  nekbone_register_backend("CUDA", cuda_init, cuda_run, cuda_finalize);
}
